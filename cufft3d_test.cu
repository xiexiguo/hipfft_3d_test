#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CHECK 1
#define VERBOSE 1

void cuda_error_check (hipError_t hipError_t, int line) {
  size_t      free, total;
  hipError_t cErr2;

  cErr2 = hipGetLastError();
  if (hipError_t != hipSuccess || cErr2 != hipSuccess) {
    printf("CUDA RT Error line %d\n", line);
    printf("CUDA RT1 Error: %s\n", hipGetErrorString(hipError_t));
    printf("CUDA RT2 Error: %s\n", hipGetErrorString(cErr2));
    hipMemGetInfo(&free,&total);
    printf("Free: %zu , Total: %zu\n", free, total);
    fflush(stdout);
    exit(-1);
  }
}


void cufft_error_check (hipfftResult_t cufftError, int line)
{
  size_t      free, total;
  hipError_t cErr2;

  cErr2 = hipGetLastError();
  if (cufftError != HIPFFT_SUCCESS || cErr2 != hipSuccess) {
    printf("CUDA FFT Error line: %d \n", line);
    switch (cufftError) {
      case HIPFFT_INVALID_PLAN:   printf("CUDA FFT1 Error (CUFFT_INVALID_PLAN)\n"); break;
      case HIPFFT_ALLOC_FAILED:   printf("CUDA FFT1 Error (CUFFT_ALLOC_FAILED)\n"); break;
      case HIPFFT_INVALID_VALUE:  printf("CUDA FFT1 Error (CUFFT_INVALID_VALUE)\n"); break;
      case HIPFFT_INTERNAL_ERROR: printf("CUDA FFT1 Error (CUFFT_INTERNAL_ERROR)\n"); break;
      case HIPFFT_EXEC_FAILED:    printf("CUDA FFT1 Error (CUFFT_EXEC_FAILED)\n"); break;
      case HIPFFT_INVALID_SIZE:   printf("CUDA FFT1 Error (CUFFT_INVALID_SIZE)\n"); break;
      default: printf("CUDA FFT1 Error (--unimplemented--) %d %d\n", cufftError, cErr2); break;
    }
    printf("CUDA FFT2 Error %s \n", hipGetErrorString(cErr2));
    hipMemGetInfo(&free,&total);
    printf("Free: %zu , Total: %zu\n", free, total);
    fflush(stdout);
    exit(-1);
  }
}


void fftcu_plan3d_z(hipfftHandle  &plan, const int *n) 
{

  hipfftResult_t cErr;

  if (VERBOSE) printf("FFT 3D (%d-%d-%d)\n", n[0], n[1], n[2]);
  cErr = hipfftPlan3d(&plan, n[2], n[1], n[0], HIPFFT_Z2Z);
  if (CHECK) cufft_error_check(cErr, __LINE__);
  //cErr = cufftSetStream(plan, cuda_stream);
  //if (CHECK) cufft_error_check(cErr, __LINE__);
}



int main()
{

  // file store data before and after fft
  FILE *f_data_in, *f_data_out;
  int n[3], fsign;
  int lmem;
  double *data_in_h, *data_out_h, *data_fft_h;
  hipfftDoubleComplex *data_d;
  hipfftHandle   plan;
  hipfftResult_t cErr;
  hipError_t  cuErr;

  // read data before and after fft
  f_data_in = fopen("data_in.dat", "r+");
  f_data_out = fopen("data_out.dat", "r+");

  // read data size
  fscanf(f_data_in,"%d %d %d\n", &n[0], &n[1], &n[2]);
  fscanf(f_data_in,"%d\n", &fsign);
  if(VERBOSE) printf("FFT 3D size: %d %d %d\n", n[0], n[1], n[2]);
  lmem = n[0] * n[1] * n[2];

  data_in_h = (double*) malloc(2*lmem*sizeof(double));
  data_out_h = (double*) malloc(2*lmem*sizeof(double));
  data_fft_h = (double*) malloc(2*lmem*sizeof(double));

  for(int i=0; i< 2*lmem; ++i)
  {
        fscanf(f_data_in,"%lf\n", &data_in_h[i]);
        fscanf(f_data_out,"%lf\n", &data_out_h[i]);
  }
  if(VERBOSE)
  {
      for(int i=0; i<10; ++i) printf("%.16e %.16e\n", data_in_h[i], data_out_h[i]);
  }

  cuErr = hipMalloc(&data_d, 2*lmem*sizeof(double));
  if(CHECK) cuda_error_check(cuErr, __LINE__);
  hipMemcpy(data_d, data_in_h, 2*lmem*sizeof(double), hipMemcpyHostToDevice);

  fftcu_plan3d_z(plan, n);
  if ( fsign < 0  ) {
    cErr = hipfftExecZ2Z(plan, data_d, data_d, HIPFFT_BACKWARD);
    if (CHECK) cufft_error_check(cErr, __LINE__);
  }
  else {
    cErr = hipfftExecZ2Z(plan, data_d, data_d, HIPFFT_FORWARD);
    if (CHECK) cufft_error_check(cErr, __LINE__);
  }

  hipMemcpy(data_fft_h, data_d, 2*lmem*sizeof(double), hipMemcpyDeviceToHost);

  for(int i=0; i<10; ++i) printf("%.16e %.16e %.16e\n", data_in_h[i], data_out_h[i],data_fft_h[i]);
  free(data_in_h);
  free(data_out_h);
  free(data_fft_h);
  fclose(f_data_in);
  fclose(f_data_out);
}
